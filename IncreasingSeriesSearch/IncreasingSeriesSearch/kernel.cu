#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <random>
#include <time.h>

#define N 1000
#define K 6


int numArray[N];
int resIndex = -1;
__device__ int dev_numArray[N];
__device__ int dev_resIndex = -1;

__global__ void FindIncSeries(int k) {

	int counter = 0;
	while (counter < k - 1 && dev_numArray[threadIdx.x + counter] < dev_numArray[threadIdx.x + counter + 1]) {
		counter++;
	}
	if (counter == k - 1)
		dev_resIndex = threadIdx.x;
}

int main() {

	srand(time(NULL));

	for (size_t i = 0; i < N; i++)
	{
		numArray[i] = rand() % 10000;
	}


	hipMemcpyToSymbol(HIP_SYMBOL(dev_numArray), numArray, N * sizeof(int));
	FindIncSeries << < 1, N - K >> > (K);
	hipError_t err = hipMemcpyFromSymbol(&resIndex, HIP_SYMBOL(dev_resIndex), sizeof(int));
	if (err != hipSuccess) {
		printf("CUDA error: %s\n", hipGetErrorString(err));
	}


	/*int index = -1;
	for (size_t i = 0; i < N - K; i++)
	{
		int counter = 0;
		while (counter < K - 1 && numArray[i + counter] < numArray[i + counter + 1]) {
			counter++;
		}
		if (counter == K - 1) {
			index = i;
		}
		else
			counter = 0;
	}*/

	/*for (size_t i = 0; i < N; i++)
	{
		printf("%d:\t%d\n", i, numArray[i]);
	}*/
	printf("Index: %d.", resIndex);
}
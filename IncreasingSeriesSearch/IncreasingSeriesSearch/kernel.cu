#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <random>
#include <time.h>

#define BLOCK_SIZE 500
#define N 10410
#define K 6


int numArray[N];
int resIndex = -1;
bool res[N - K];
__device__ int dev_numArray[N];
__device__ int dev_resIndex = -1;
__device__ bool dev_res[N - K];

__global__ void FindIncSeries_N(int k) {

	int x = threadIdx.x;
	int counter = 0;

	while (counter < k - 1 && dev_numArray[x + counter] < dev_numArray[x + counter + 1]) {
		counter++;
	}
	if (counter == k - 1)
		dev_resIndex = x;
}

__global__ void FindIncSeries_N_BLOCKS(int k) {

	__shared__ int shr_numArray[N];
	int x = threadIdx.x;
	int bl_x = blockIdx.x;
	int counter = 0;

	int i = x + bl_x * blockDim.x;
	if (i < N)
		shr_numArray[i] = dev_numArray[i];

	//túlindex védelem
	if (bl_x == N / blockDim.x + 1 && x > N % blockDim.x - k)
		return;

	while (counter < k - 1 && shr_numArray[bl_x * blockDim.x + x + counter] < shr_numArray[bl_x * blockDim.x + x + counter + 1]) {
		counter++;
	}
	if (counter == k - 1)
		dev_resIndex = bl_x * BLOCK_SIZE + x;
}

__global__ void FindIncSeries_OneCicle() {
	int x = threadIdx.x;
	int y = threadIdx.y;
	if (y == 0)
		dev_res[x] = true;

	if (!(dev_numArray[x + y] < dev_numArray[x + y + 1])) //nem növekvő
		dev_res[x] = false;
}

int main() {

	//random generate
	srand(time(NULL));
	for (size_t i = 0; i < N; i++)
	{
		numArray[i] = rand() % 10000;
	}
	numArray[10400] = 1;
	numArray[10401] = 2;
	numArray[10402] = 3;
	numArray[10403] = 4;
	numArray[10404] = 5;
	numArray[10405] = 6;
	numArray[10406] = 7;
	numArray[10407] = 8;
	numArray[10408] = 9;
	numArray[10409] = 10;


	hipMemcpyToSymbol(HIP_SYMBOL(dev_numArray), numArray, N * sizeof(int));
	//FindIncSeries_N << < 1, N - K >> > (K);
	FindIncSeries_N_BLOCKS << < N / BLOCK_SIZE + 1, BLOCK_SIZE >> > (K);
	hipMemcpyFromSymbol(&resIndex, HIP_SYMBOL(dev_resIndex), sizeof(int));

	//FindIncSeries_OneCicle << < 1, dim3(N - K, K - 1) >> > ();
	hipError_t err = hipGetLastError();
	if (err != hipSuccess) {
		printf("CUDA error after kernel launch: %s\n", hipGetErrorString(err));
		return -1;
	}
	//hipMemcpyFromSymbol(res, HIP_SYMBOL(dev_res), (N - K) * sizeof(bool));

	//CPU
	/*int index = -1;
	for (size_t i = 0; i < N - K; i++)
	{
		int counter = 0;
		while (counter < K - 1 && numArray[i + counter] < numArray[i + counter + 1]) {
			counter++;
		}
		if (counter == K - 1) {
			index = i;
		}
		else
			counter = 0;
	}*/

	//printing the nums
	/*for (size_t i = 0; i < N; i++)
	{
		printf("%d:\t%d\n", i, numArray[i]);
	}*/
	/*for (size_t i = 0; i < N - K; i++)
	{
		if (res[i] == true)
		{
			printf("Ez egy jo index: %d\n", i);
		}
	}*/
	printf("Index, ahol %d szam novekvo sorrendben van egymas utan: %d.", K, resIndex);
}

#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <random>
#include <vector>
#define N 10000
#define BLOCK_SIZE 512


__device__ int dev_minIdx;

__global__ void MinimumMulFind_N(int* dev_v1) {
	int x = blockIdx.x * BLOCK_SIZE + threadIdx.x;
	__shared__ size_t min;


	if (x == 0)
		min = SIZE_MAX;
	else if (x > N)
		return;
	__syncthreads();

	size_t mul = 1;
	for (size_t i = 0; i < 10; i++)
	{
		mul *= dev_v1[x + i];
	}

	if (atomicMin(&min, mul) > mul)
		atomicExch(&dev_minIdx, x);
}


int main() {
	//random generátor setup
	std::random_device dev;
	std::mt19937 gen(dev());
	std::uniform_int_distribution<>dist(1, 10);

	//feltöltés
	std::vector<int> v1;
	for (size_t i = 0; i < N; i++)
		v1.push_back(dist(gen));


	//CPU
	size_t min = SIZE_MAX;
	int minIdx = -1;
	for (size_t i = 0; i < N - 10; i++)
	{
		size_t current = 1;
		for (size_t j = 0; j < 10; j++)
			current *= v1[i + j];

		if (min > current) {
			min = current;
			minIdx = i;
		}
	}

	//GPU

	//memory allocation
	int* dev_v1;
	int GPU_minIdx;

	hipEvent_t start_event, end_event;
	hipEventCreate(&start_event);
	hipEventCreate(&end_event);


	hipMalloc((void**)&dev_v1, N * sizeof(int));
	hipError_t err = hipGetLastError();

	//memory copying

	hipMemcpy(dev_v1, &v1[0], N * sizeof(int), hipMemcpyHostToDevice);
	err = hipGetLastError();

	hipEventRecord(start_event, 0);
	MinimumMulFind_N << <N / BLOCK_SIZE + 1, BLOCK_SIZE >> > (dev_v1);
	hipEventRecord(end_event, 0);

	hipEventSynchronize(start_event);
	hipEventSynchronize(end_event);
	hipDeviceSynchronize();
	err = hipGetLastError();

	hipMemcpyFromSymbol(&GPU_minIdx, HIP_SYMBOL(dev_minIdx), sizeof(int));
	err = hipGetLastError();

	hipFree(dev_v1);


	float elapsed_ms;
	hipEventElapsedTime(&elapsed_ms, start_event, end_event);


	/*for (size_t i = 0; i < N; i++)
	{
		printf("%llu: %d\n", i, v1[i]);
	}*/
	printf("\n%d", minIdx);
	printf("\n%d", GPU_minIdx);


	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, 0);
	int maxThreads = deviceProp.reg;

	printf("\n%d", maxThreads);
}
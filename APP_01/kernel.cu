﻿
#include "hip/hip_runtime.h"


#include <stdio.h>

int tomb[] = { 2,4,8,16,32 };

__device__ int dev_tomb[5];

__global__ void Szorzas(int num)
{
	int i = threadIdx.x;

	dev_tomb[i] *= num;
}

int main()
{
	hipMemcpyToSymbol(HIP_SYMBOL(dev_tomb), tomb, 5 * sizeof(int));
	Szorzas <<< 1, 5 >>> (3);
	hipMemcpyFromSymbol(tomb, HIP_SYMBOL(dev_tomb), 5 * sizeof(int));
    return 0;
}

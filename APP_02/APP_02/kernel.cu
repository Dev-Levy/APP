#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

char word[] = "all";
char sentence[] = "it’s all a matter of all perspective";
const int w_len = 3;
const int s_len = 36;
int res = -2;

bool results[s_len];


__device__ char dev_word[w_len];
__device__ char dev_sentence[s_len];
__device__ int dev_w_len;
__device__ int dev_s_len;
__device__ int dev_res;

__device__ bool dev_results[s_len];


__global__ void FindWord_1_GPU_CORE()
{
	dev_res = -1;

	for (int i = 0; i <= dev_s_len - dev_w_len; i++)
	{
		int j = 0;

		while (dev_sentence[i + j] == dev_word[j] && j < dev_w_len)
			j++;

		if (j == dev_w_len)
			dev_res = i;
	}
}
__global__ void FindWord_N_GPU_CORE()
{
	dev_res = -1;
	int i = threadIdx.x;
	int j = 0;

	while (dev_sentence[i + j] == dev_word[j] && j < dev_w_len)
		j++;

	if (j == dev_w_len)
		dev_res = i;
}
__global__ void FindWord_NxM_GPU_CORE() 
{
	dev_results[threadIdx.x] = true;

	if (dev_sentence[threadIdx.x + threadIdx.y] != dev_word[threadIdx.y])
		dev_results[threadIdx.x] = false;
}

int main()
{
	hipMemcpyToSymbol(HIP_SYMBOL(dev_word), word, sizeof(dev_word));
	hipMemcpyToSymbol(HIP_SYMBOL(dev_sentence), sentence, sizeof(dev_sentence));
	hipMemcpyToSymbol(HIP_SYMBOL(dev_w_len), &w_len, sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(dev_s_len), &s_len, sizeof(int));


	//FindWord_1_GPU_CORE << <1, 1 >> > ();
	//FindWord_N_GPU_CORE << <1, s_len - w_len + 1 >> > ();
	FindWord_NxM_GPU_CORE << <1, dim3(s_len, w_len) >> > ();


	hipMemcpyFromSymbol(results, HIP_SYMBOL(dev_results), sizeof(dev_results));


	printf("%d", results);

	return 0;
}

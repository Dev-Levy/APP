#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

char* word = "ASD";
char* sentence = "FGHASDJKLASDGHJKASD";
int w_len = 3;
int s_len = 19;
int res = -1;

__device__ char* dev_word;
__device__ char* dev_sentence;
__device__ int dev_w_len;
__device__ int dev_s_len;
__device__ int dev_res;

__global__ void FindWord() 
{
	int found = -1;

	for (size_t i = 0; i < dev_s_len; i++)
	{
		int j = 0;
		while (dev_sentence[i + j] == dev_word[j] && j < dev_w_len)
			j++;

		if (j == dev_w_len)
			found = i;
	}
}

int main()
{
	hipMemcpyToSymbol(HIP_SYMBOL(dev_word), word, sizeof(word));
	hipMemcpyToSymbol(HIP_SYMBOL(dev_sentence), sentence, sizeof(word));
	hipMemcpyToSymbol(HIP_SYMBOL(dev_w_len), &w_len, sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(dev_s_len), &s_len, sizeof(int));

	FindWord << <1, 1 >> > ();

	hipMemcpyFromSymbol(&res, HIP_SYMBOL(dev_res), sizeof(int));

	printf("%d", res);
	
	return 0;
}
